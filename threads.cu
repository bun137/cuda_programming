#include "hip/hip_runtime.h"


#include <stdio.h>

// Device Code
__global__ void print_thread() {
  printf("x: %d  y: %d  z: %d \n", threadIdx.x, threadIdx.y, threadIdx.z);
}

// Host code
int main() {
  // kernel launch parameters

  dim3 block(2, 1);
  dim3 grid(2, 2);

  print_thread<<<grid, block>>>(); // async call
  printf("Hello from CPU \n");
  hipDeviceSynchronize(); // will make the prgram stall till all the launched
                           // kernels have finished execution

  hipDeviceReset();
  return 0;
}
