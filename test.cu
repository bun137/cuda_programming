#include "hip/hip_runtime.h"

#include <stdio.h>

__global__ void helloi() { printf("hello cuda\n"); }

int main() {
  helloi<<<1, 10>>>();
  hipDeviceSynchronize();
  hipDeviceReset();
  return 0;
}
