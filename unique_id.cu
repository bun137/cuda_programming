
#include "hip/hip_runtime.h"


#include <stdio.h>

// Device Code
__global__ void print_thread() {
  printf("x: %d  y: %d  z: %d \n", threadIdx.x, threadIdx.y, threadIdx.z);
  int t = threadIdx.x;
  int off = blockIdx.x * blockDim.x;
  int gid = t + off;
  printf("gid: %d \n", gid);
}

// Host code
int main() {
  // kernel launch parameters
  int a[10];
  int d[] = {1, 2, 3, 4, 5, 6, 7, 8, 9, 10};
  int d1[] = {11, 12, 13, 14, 15, 16, 17, 18, 19, 20};

  int size = sizeof(a) / sizeof(int);
  for (int i = 0; i < size; i++) {
    a[i] = d[i] + d1[i];
  }

  dim3 block(4, 1);
  dim3 grid(2, 1);

  print_thread<<<grid, block>>>(); // async call
  printf("Hello from CPU \n");
  hipDeviceSynchronize(); // will make the prgram stall till all the launched
                           // kernels have finished execution

  hipDeviceReset();
  return 0;
}
